#include <hipblas.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "utils.h"

float testCuBLASPad(int M, int N, int iters, int warmup) {
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  float* A;
  float* B;
  float* C;

  auto op_a = HIPBLAS_OP_N;
  auto op_b = HIPBLAS_OP_N;

  int lda = M;
  int ldb = M;
  int ldc = M;

  CUDA_CHECK(hipMalloc((void**)&A, M * M * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&B, M * N * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&C, M * N * sizeof(float)));

  auto runner = [&]() {
    float time = 0;
    for (int i = 0; i < iters; ++i) {
      hipEvent_t start, end;
      float elapsed = 0;

      // Timing info
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      const float alpha = 1.0;
      const float beta = 0.0;

      hipblasStatus_t  cublas_result = hipblasSgemm(cublas_handle,
						  op_a, op_b,
						  M, N, M,
						  &alpha,
						  A, lda,
						  B, ldb,
						  &beta,
						  C, ldc);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      time += elapsed;
      assert(cublas_result == HIPBLAS_STATUS_SUCCESS);
    }
    return (time / iters);
  };

  if (warmup) { runner(); }
  float time = runner();

  CUDA_CHECK(hipFree((void*)A));
  CUDA_CHECK(hipFree((void*)B));
  CUDA_CHECK(hipFree((void*)C));

  return time;
}


float testCuBLASNoPad(int M, int N, int iters, int warmup) {
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  float* A;
  float* B;
  float* C;

  auto op_a = HIPBLAS_OP_N;

  int lda = M;
  int ldb = M;
  int ldc = M;

  CUDA_CHECK(hipMalloc((void**)&A, M * M * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&B, M * N * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&C, M * N * sizeof(float)));

  auto runner = [&]() {
    float time = 0;
    for (int i = 0; i < iters; ++i) {
      hipEvent_t start, end;
      float elapsed = 0;

      // Timing info
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      const float alpha = 1.0;

      hipblasStatus_t cublas_result = hipblasStrmm(cublas_handle,
						 HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
						 op_a, HIPBLAS_DIAG_NON_UNIT,
						 M, N,
						 &alpha,
						 A, lda,
						 B, ldb,
						 C, ldc);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      time += elapsed;
      assert(cublas_result == HIPBLAS_STATUS_SUCCESS);
    }
    return (time / iters);
  };

  if (warmup) { runner(); }
  float time = runner();

  CUDA_CHECK(hipFree((void*)A));
  CUDA_CHECK(hipFree((void*)B));
  CUDA_CHECK(hipFree((void*)C));

  return time;
}

int main(int argc, char *argv[]) {
  int M = std::stoi(argv[1]);
  int N = std::stoi(argv[2]);
  bool pad = (bool)(std::stoi(argv[3]));
  int iters = std::stoi(argv[4]);
  int warmup = std::stoi(argv[5]);

  if (pad) {
    float time = testCuBLASPad(M, N, iters, warmup);
    std::cout << "RESULTS," << time << std::endl;
  } else {
    float time = testCuBLASNoPad(M, N, iters, warmup);
    std::cout << "RESULTS," << time << std::endl;
  }
}
